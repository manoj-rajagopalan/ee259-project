#include "hip/hip_runtime.h"
#include "AssimpOptixRayTracer.hpp"
#include "cuda_optix_sentinel.h"

// https://forums.developer.nvidia.com/t/linking-optix7-with-gcc/107036
#include <optix_function_table_definition.h>

#include <array>
#include <cassert>
#include <fstream>
#include <iterator>
#include <thread>

#include "LaunchParams.hpp"
#include "RaiiScopeLimitsLogger.hpp"

namespace {

// Help OptiX runtime route error messages to
manojr::AssimpOptixRayTracer *g_theAssimpOptixRayTracer = nullptr;

void optixLogCallback(unsigned int level, const char *tag, const char *msg, void* /* cbData */)
{
	g_theAssimpOptixRayTracer->logOptixMessage(level, tag, msg);
}

} // namespace

namespace manojr
{

AssimpOptixRayTracer::AssimpOptixRayTracer(
    std::mutex& rayTracingCommandMutex,
    std::condition_variable& rayTracingCommandConditionVariable,
    int& rayTracingCommand
)
: rayTracingCommandMutex_(rayTracingCommandMutex),
  rayTracingCommandConditionVariable_(rayTracingCommandConditionVariable),
  rayTracingCommand_(rayTracingCommand),
  executionFrame_{ 0 }
{
    // Capture all OptiX messages via optixLogCallback() above.
    g_theAssimpOptixRayTracer = this;
}

void AssimpOptixRayTracer::eventLoop()
{
    initialize();
    bool quit = false;
    while(!false) {
        std::unique_lock<std::mutex> lock(rayTracingCommandMutex_);
        rayTracingCommandConditionVariable_.wait(lock, [](){ rayTracingCommand_ != 0; });
        if(rayTracingCommand_ < 0) { // quit
            rayTracingCommand_ = 0;
            quit = true;
        }
        else if(rayTracingCommand_ == 1) { // ray-trace
            rayTracingCommand_ = 0;
            lock.unlock();
            runRayTracing(); // TODO capture result
        }
        else if(rayTracingCommand_ == 2) { // new scene
            setScene(scene);
            aiMatrix4x4 identityTransform{};
            setSceneTransform(identityTransform);
            rayTracingCommand_ = 0;
            lock.unlock();
            runRayTracing(); // TODO capture result
        }
    }
    finalize();
}

void AssimpOptixRayTracer::initialize()
{
    ExecutionCursor whereInProgram;
    whereInProgram.frame = std::to_string(executionFrame_++);
    RaiiScopeLimitsLogger scopeLog(logInfo_, whereInProgram, __func__);

    initializeCuda_(whereInProgram.advance());
    initializeOptix_(whereInProgram.advance());
}

AssimpOptixRayTracer::~AssimpOptixRayTracer()
{
    ExecutionCursor whereInProgram;
    whereInProgram.frame = std::to_string(executionFrame_++);
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, "AssimpOptixRayTracer DTOR");
    finalizeOptix_(whereInProgram.advance());
    finalizeCuda_(whereInProgram.advance());
}

void AssimpOptixRayTracer::registerLoggingFunctions(
    std::function<void(std::string const&)> logInfo,
    std::function<void(std::string const&)> logError
)
{
    logInfo_ = logInfo;
    logError_ = logError;
}

void AssimpOptixRayTracer::initializeCuda_(ExecutionCursor whereInProgram)
{
    try {
        int cudaDeviceCount = 0;
        CUDA_CHECK( GetDeviceCount(&cudaDeviceCount) );
        logInfo_('[' + whereInProgram.toString() + "] "
                 + "Found " + std::to_string(cudaDeviceCount) + " CUDA device");
        assert(cudaDeviceCount == 1);

        int cudaDevice = -1;
        CUDA_CHECK( GetDevice(&cudaDevice) );
        hipDeviceProp_t cudaDeviceProps;
        CUDA_CHECK( GetDeviceProperties(&cudaDeviceProps, cudaDevice) );
        logInfo_('[' + whereInProgram.toString() + "] "
                 + std::string("CUDA device name is ") + cudaDeviceProps.name);

        CUDA_CHECK( SetDevice(cudaDevice) );

        CUDA_CHECK( StreamCreate(&cudaStream_) );
        modelVertexBufferOnGpu_.cudaStream = cudaStream_;
        worldVertexBufferOnGpu_.cudaStream = cudaStream_;
        indexBufferOnGpu_.cudaStream = cudaStream_;
        rayHitVerticesOnGpu_.cudaStream = cudaStream_;
        gasBuild_.cudaStream = cudaStream_;
    }
    catch(std::exception& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::finalizeCuda_(ExecutionCursor whereInProgram)
{
    try {
        CUDA_CHECK( StreamDestroy(cudaStream_) );
    }
    catch(std::exception& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}


void AssimpOptixRayTracer::logOptixMessage(unsigned int level, const char *tag, const char *msg)
{
    std::ostringstream s;
	s << "-OptiX @" << level << '[' << tag << "] " << msg;
	logInfo_(s.str());
}


/// @brief Push geometry into GPU and reserve memory for results.
void AssimpOptixRayTracer::setScene(aiScene const& scene)
{
    ExecutionCursor whereInProgram;
    whereInProgram.frame = std::to_string(executionFrame_++);
    RaiiScopeLimitsLogger raiiScopeLogger(logInfo_, whereInProgram, __func__);

    if(!scene.HasMeshes()) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** Scene has no meshes!");
        return;
    }
    if(scene.mNumMeshes != 1) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** Require scene to have 1 mesh. "
                "Has " + std::to_string(scene.mNumMeshes));
        return;
    }
    aiMesh const& mesh = *scene.mMeshes[0];
    if(mesh.mPrimitiveTypes != aiPrimitiveType_TRIANGLE) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** Require mesh to have only triangles."
                " Not the case");
        return;
    }

    // - Vertex and index buffers for geometry - 

    try {
        modelVertexBufferOnGpu_.asyncAllocAndUpload(mesh.mVertices, mesh.mNumVertices);
        worldVertexBufferOnGpu_.asyncAlloc(mesh.mNumVertices * sizeof(aiVector3D));
        worldVertexBufferOnGpu_.numElements = mesh.mNumVertices;
        worldVertexBufferOnGpu_.sizeOfElement = sizeof(aiVector3D);

        std::vector<int16_t> indexBuffer(mesh.mNumFaces * 3);
        int indexCounter = 0;
        for(uint32_t i = 0; i < mesh.mNumFaces; ++i) {
            const aiFace& face = mesh.mFaces[i];
            assert(face.mNumIndices == 3);
            assert(face.mIndices[0] < (1u << 16));
            indexBuffer[indexCounter++] = (uint16_t) face.mIndices[0];
            assert(face.mIndices[1] < (1u << 16));
            indexBuffer[indexCounter++] = (uint16_t) face.mIndices[1];
            assert(face.mIndices[2] < (1u << 16));
            indexBuffer[indexCounter++] = (uint16_t) face.mIndices[2];
        }
        indexBufferOnGpu_.asyncAllocAndUpload(indexBuffer);
    }
    catch(std::exception& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

__device__ __forceinline__
float inner_prod(float4 const& m, float3 const& v )
{
    return m.x*v.x + m.y*v.y + m.z*v.z + m.w /* *1 */;
}

__global__
void transformVertexBufferOnGpu(void *worldVertices,
                                void const *modelVertices,
                                int32_t const numVertices,
                                void const *modelToWorldTransform /* 3x4, row-major*/)
{
    int const linearIdx = blockIdx.x * blockDim.x + threadIdx.x;
    float3& w = *(((float3*) worldVertices) + linearIdx);
    float3 const& v = *(((float3 const*) modelVertices) + linearIdx);
    float4 const *const M = (float4 const *) modelToWorldTransform;
    w.x = inner_prod(M[0], v);
    w.y = inner_prod(M[1], v);
    w.z = inner_prod(M[2], v);
}

void AssimpOptixRayTracer::setSceneTransform(aiMatrix4x4 const& modelToWorldTransform)
{
    ExecutionCursor whereInProgram;
    whereInProgram.frame = std::to_string(executionFrame_++);
    RaiiScopeLimitsLogger raiiScopeLogger(logInfo_, whereInProgram, __func__);

    try {
        AsyncCudaBuffer modelToWorldTransformOnGpu{cudaStream_};
        modelToWorldTransformOnGpu.asyncAllocAndUpload((float const*) &modelToWorldTransform, 12);
        // 8 block with 32 threads each
        int const kNumThreadsPerBlock = 32;
        // TODO: large meshes can exceed CUDA limits
        int const kNumBlocks = (modelVertexBufferOnGpu_.numElements + kNumThreadsPerBlock-1) / kNumThreadsPerBlock;
        transformVertexBufferOnGpu<<<kNumBlocks, kNumThreadsPerBlock, 0, cudaStream_>>>(
            worldVertexBufferOnGpu_.d_pointer(),
            modelVertexBufferOnGpu_.d_pointer(),
            modelVertexBufferOnGpu_.numElements,
            modelToWorldTransformOnGpu.d_pointer()
        );
        CUDA_CHECK( GetLastError() );

        // Rebuild BVH etc.
        buildOptixAccelerationStructures_(whereInProgram.advance()); // set gasBuild_, gasHandle_

    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::setTransmitter(manojr::Transmitter const& transmitter) {
    transmitter_ = transmitter;

    // Allocate on-GPU memory for ray-tracing results (collision points in world space).
    // In worst-case all rays will hit geometry.
    int32_t const numRays = transmitter_.numRays_x * transmitter_.numRays_y;
    rayHitVerticesOnGpu_.asyncFree();
    rayHitVerticesOnGpu_.asyncAlloc(numRays * sizeof(float3));
    // The following metadata needs to be ready before we 'download' results from GPU to CPU.
    rayHitVerticesOnGpu_.sizeOfElement = sizeof(float3);
    // Haven't ray-traced yet.
    // Will set this to the value of atomic counter (incremented by ray-collisions), post-ray-tracing.
    rayHitVerticesOnGpu_.numElements = 0;
}

void AssimpOptixRayTracer::setTransmitterTransform(aiMatrix4x4 const& transmitterToWorldTransform /* 3x3 rotation matrix only! */)
{
    // Though the name mentions 'transform', this really is only a 3x3 rotation.
    // The AssImp Qt Viewer library which provides our GLView implementation is sloppy with
    // using 4x4 matrices to maintain 3x3 rotation, presumably because AssImp library itself
    // has no notion of 3x3 matrices.

    // The unit-vectors for the transmitter in world-space are simply the columns of the
    // transmitter-to-world rotation matrix.

	transmitter_.xUnitVector.x = transmitterToWorldTransform[0][0];
	transmitter_.xUnitVector.y = transmitterToWorldTransform[1][0];
	transmitter_.xUnitVector.z = transmitterToWorldTransform[2][0];
	
	transmitter_.yUnitVector.x = transmitterToWorldTransform[0][1];
	transmitter_.yUnitVector.y = transmitterToWorldTransform[1][1];
	transmitter_.yUnitVector.z = transmitterToWorldTransform[2][1];
	
	transmitter_.zUnitVector.x = transmitterToWorldTransform[0][2];
	transmitter_.zUnitVector.y = transmitterToWorldTransform[1][2];
	transmitter_.zUnitVector.z = transmitterToWorldTransform[2][2];
}

aiScene* AssimpOptixRayTracer::runRayTracing()
{
    ExecutionCursor whereInProgram;
    whereInProgram.frame = std::to_string(executionFrame_++);
    RaiiScopeLimitsLogger raiiScopeLogger(logInfo_, whereInProgram, __func__);

    aiScene *const pointCloudScene = new aiScene;
    try {
        OptixLaunchParams launchParams{};
        {
            launchParams.pointCloud = (void*) rayHitVerticesOnGpu_.d_pointer();
            launchParams.gasHandle = gasHandle_;
            launchParams.transmitter = transmitter_;
            launchParams.gpuAtomicNumHits = 0;
        }
        AsyncCudaBuffer launchParamsOnGpu{cudaStream_};
        launchParamsOnGpu.asyncAllocAndUpload(&launchParams, 1);
        OPTIX_CHECK(
            optixLaunch(optixPipeline_,
                        cudaStream_,
                        (hipDeviceptr_t) launchParamsOnGpu.d_pointer(),
                        launchParamsOnGpu.sizeInBytes,
                        &optixSbt_,
                        launchParams.transmitter.numRays_x,
                        launchParams.transmitter.numRays_y,
                        1)
        );
        CUDA_CHECK( StreamSynchronize(cudaStream_) );
        launchParamsOnGpu.asyncDownload(&launchParams, 1);
        launchParamsOnGpu.sync(); // Ensure download completes before accessing results.
        rayHitVerticesOnGpu_.numElements = launchParams.gpuAtomicNumHits;
        
        aiMesh *const pointCloudMesh = new aiMesh;
        pointCloudMesh->mNumVertices = launchParams.gpuAtomicNumHits;
        pointCloudMesh->mVertices = new aiVector3D[launchParams.gpuAtomicNumHits];
        rayHitVerticesOnGpu_.asyncDownload(pointCloudMesh->mVertices, pointCloudMesh->mNumVertices);
        rayHitVerticesOnGpu_.sync(); // Ensure download completes before accessing results.

        using aiMeshPtr_t = aiMesh*;
        pointCloudScene->mMeshes = new aiMeshPtr_t[1];
        pointCloudScene->mMeshes[0] = pointCloudMesh;
        pointCloudScene->mNumMeshes = 1;

        // Scenes must have a root node which indexes into the master mesh-array within the parent scene.
        aiNode *const pointCloudNode = new aiNode;
        pointCloudNode->mName = "point-cloud";
        pointCloudNode->mNumMeshes = 1;
        pointCloudNode->mMeshes = new unsigned int [pointCloudNode->mNumMeshes];
        pointCloudNode->mMeshes[0] = 0; // First entry in the array of meshes in the scene.
        pointCloudNode->mNumChildren = 0;
        pointCloudNode->mChildren = nullptr;
        pointCloudScene->mRootNode = pointCloudNode;
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }

    return pointCloudScene;
}

void AssimpOptixRayTracer::createOptixDeviceContext_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    try {
        OptixDeviceContextOptions optixDeviceContextOptions;
        optixDeviceContextOptions.logCallbackFunction = &optixLogCallback;
        optixDeviceContextOptions.logCallbackLevel = 4;
        optixDeviceContextOptions.validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL;
        OPTIX_CHECK(
            optixDeviceContextCreate(cuCtx_, &optixDeviceContextOptions, &optixDeviceContext_)
        );
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::buildOptixAccelerationStructures_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger execLog(logInfo_, whereInProgram, __func__);

	// https://raytracing-docs.nvidia.com/optix7/api/struct_optix_accel_build_options.html
	OptixAccelBuildOptions optixAccelBuildOptions = {};
	{
		// https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ggaff328b8278fbd1900558593599698bbaafa820662dca4a85935ab74c704665d93
		optixAccelBuildOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_RANDOM_VERTEX_ACCESS;

		// https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga411c1c6d9f4d8e039ae19e9dea65958a
		optixAccelBuildOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

		// Ignore optixAccessBuildOptions.motionOptions
	}

	// https://raytracing-docs.nvidia.com/optix7/api/struct_optix_build_input.html
	OptixBuildInput optixBuildInput = {};
	{
		// https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga7932d1d9cdf33506a75a5da5d8a62d94
		optixBuildInput.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

		// https://raytracing-docs.nvidia.com/optix7/api/struct_optix_build_input_triangle_array.html
        OptixBuildInputTriangleArray& triangleArray = optixBuildInput.triangleArray;
		triangleArray.vertexBuffers = (hipDeviceptr_t*) &(worldVertexBufferOnGpu_.d_ptr);
		triangleArray.numVertices = (unsigned int) worldVertexBufferOnGpu_.numElements;
		// https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga55c6d96161ef202d48023a8a1d126102
		triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
        triangleArray.vertexStrideInBytes = 0; // tightly-packed

		triangleArray.indexBuffer = (hipDeviceptr_t) indexBufferOnGpu_.d_pointer();
		triangleArray.numIndexTriplets = (unsigned int) indexBufferOnGpu_.numElements / 3; // triplets, remember?
		// https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gaa98b8fb6bf2d2455c310125f3fab74e6
		triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_SHORT3;
		triangleArray.indexStrideInBytes = 0; // tightly-packed

        const unsigned int triangle_flags[]{OPTIX_GEOMETRY_FLAG_NONE};
        triangleArray.flags = triangle_flags;
        triangleArray.numSbtRecords = 1;
	}

    // Compute buffer sizes for building acceleration structures
	OptixAccelBufferSizes optixAccelBufferSizes{};
	OPTIX_CHECK(
		optixAccelComputeMemoryUsage(optixDeviceContext_,
		                             &optixAccelBuildOptions,
									 &optixBuildInput, 1,
									 &optixAccelBufferSizes)
	);
    assert(optixAccelBufferSizes.tempUpdateSizeInBytes == 0);

    // Allocate buffers for the build
    gasBuild_.asyncResize(optixAccelBufferSizes.outputSizeInBytes);

    AsyncCudaBuffer gasBuildTemp{cudaStream_};
    gasBuildTemp.asyncResize(optixAccelBufferSizes.tempSizeInBytes);

    OPTIX_CHECK(
        optixAccelBuild(optixDeviceContext_,
                        cudaStream_,
                        &optixAccelBuildOptions,
                        &optixBuildInput, 1,
                        (hipDeviceptr_t) gasBuildTemp.d_pointer(), gasBuildTemp.sizeInBytes,
                        (hipDeviceptr_t) gasBuild_.d_pointer(), gasBuild_.sizeInBytes,
                        &gasHandle_,
                        nullptr, 0 // emitted properties
                        );
    );
}

std::string AssimpOptixRayTracer::loadOptixModuleSourceCode_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    std::string sourceCode;
    try {
        std::ifstream sourceCodeFile("ee259.ptx");
        // https://stackoverflow.com/a/2912614
        std::istreambuf_iterator<char> sourceCodeFileIter{sourceCodeFile};
        sourceCode.assign(sourceCodeFileIter,
                          std::istreambuf_iterator<char>{});
        logInfo_('[' + whereInProgram.toString() + "] Source code has "
                 + std::to_string(sourceCode.size()) + " chars");
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
    return sourceCode;
}

void AssimpOptixRayTracer::makeOptixModule_(OptixPipelineCompileOptions optixPipelineCompileOptions,
                                            char* logBuffer,
                                            size_t logBufferSize,
                                            ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    try {
        ExecutionCursor whereInSubProgram = whereInProgram.nextLevel();

        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_module_compile_options.html
        OptixModuleCompileOptions optixModuleCompileOptions{};
        {
            optixModuleCompileOptions.maxRegisterCount = 0; // unlimited
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gaea8ecab8ad903804364ea246eefc79b2
            optixModuleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0; // none, for debug
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga2a921efc5016b2b567fa81ddb429e81a
            optixModuleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
            optixModuleCompileOptions.numBoundValues = 0;
            optixModuleCompileOptions.boundValues = 0; // null pointer
            optixModuleCompileOptions.numPayloadTypes = 0;
            optixModuleCompileOptions.payloadTypes = 0;
        }

        std::string optixModuleSourceCode = loadOptixModuleSourceCode_(whereInSubProgram.advance());
        OPTIX_CHECK(
            optixModuleCreate(optixDeviceContext_,
                              &optixModuleCompileOptions,
                              &optixPipelineCompileOptions,
                              optixModuleSourceCode.c_str(),
                              optixModuleSourceCode.size(),
                              logBuffer, &logBufferSize,
                              &optixModule_)
        );
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

OptixPipelineCompileOptions AssimpOptixRayTracer::makeOptixPipelineCompileOptions_()
{
    // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_pipeline_compile_options.html
    OptixPipelineCompileOptions optixPipelineCompileOptions;
    optixPipelineCompileOptions.usesMotionBlur = 0;
    // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gabd8bb7368518a44361e045fe5ad1fd17
    optixPipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_ANY;
    optixPipelineCompileOptions.numPayloadValues = 0; // TODO
    optixPipelineCompileOptions.numAttributeValues = 2; // min
    // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga95e8175699d1a23c5c1d5333c4468190
    optixPipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_STACK_OVERFLOW |
                                                 OPTIX_EXCEPTION_FLAG_TRACE_DEPTH |
                                                 OPTIX_EXCEPTION_FLAG_USER |
                                                 OPTIX_EXCEPTION_FLAG_DEBUG;
    optixPipelineCompileOptions.pipelineLaunchParamsVariableName = "launchParams";
    // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga1171b332da08991dd9e6ef54b52b3ba4
    optixPipelineCompileOptions.usesPrimitiveTypeFlags = OPTIX_PRIMITIVE_TYPE_FLAGS_TRIANGLE;
    optixPipelineCompileOptions.allowOpacityMicromaps = 0u;
    return optixPipelineCompileOptions;
}

OptixPipelineLinkOptions AssimpOptixRayTracer::makeOptixPipelineLinkOptions_()
{
    // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_pipeline_link_options.html
    OptixPipelineLinkOptions optixPipelineLinkOptions;
    optixPipelineLinkOptions.maxTraceDepth = 5;
    return optixPipelineLinkOptions;
}

void AssimpOptixRayTracer::makeRaygenProgramGroup_(char *logBuffer,
                                                   size_t logBufferSize,
                                                   ExecutionCursor whereInProgram)
{
    // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga0b44fd0708cced8b77665cfac5453573

    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    try {
        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_desc.html
        OptixProgramGroupDesc raygenProgramGroupDescription{};
        {
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gabca35b1218b4df575a5c42926da0d978
            raygenProgramGroupDescription.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gaa65e764f5bba97fda45d4453c0464596
            raygenProgramGroupDescription.flags = OPTIX_PROGRAM_GROUP_FLAGS_NONE;

            // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_single_module.html
            raygenProgramGroupDescription.raygen.module = optixModule_;
            raygenProgramGroupDescription.raygen.entryFunctionName = "__raygen__rg";
        }

        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_options.html
        OptixProgramGroupOptions raygenProgramGroupOptions{}; // no payload

        // https://raytracing-docs.nvidia.com/optix7/api/optix__host_8h.html#aa3515445a876a8a381ced002e4020d42
        OPTIX_CHECK(
            optixProgramGroupCreate(optixDeviceContext_,
                                    &raygenProgramGroupDescription, 1, // array
                                    &raygenProgramGroupOptions,
                                    logBuffer, &logBufferSize, // modified
                                    &raygenProgramGroup_)
        );
        logBuffer[logBufferSize] = '\0';
        logInfo_('[' + whereInProgram.toString() + "] " + (char*) logBuffer);
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::makeHitGroupProgramGroup_(char *logBuffer,
                                                     size_t logBufferSize,
                                                     ExecutionCursor whereInProgram)
{
    // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga0b44fd0708cced8b77665cfac5453573

    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    try {
        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_desc.html
        OptixProgramGroupDesc hitGroupProgramGroupDescription;
        {
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gabca35b1218b4df575a5c42926da0d978
            hitGroupProgramGroupDescription.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gaa65e764f5bba97fda45d4453c0464596
            hitGroupProgramGroupDescription.flags = OPTIX_PROGRAM_GROUP_FLAGS_NONE;

            // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_hitgroup.html
            hitGroupProgramGroupDescription.hitgroup.moduleCH = optixModule_;
            hitGroupProgramGroupDescription.hitgroup.entryFunctionNameCH = "__closesthit__xpoint";
        }

        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_options.html
        OptixProgramGroupOptions hitGroupProgramGroupOptions{}; // no payload

        // https://raytracing-docs.nvidia.com/optix7/api/optix__host_8h.html#aa3515445a876a8a381ced002e4020d42
        OPTIX_CHECK(
            optixProgramGroupCreate(optixDeviceContext_,
                                    &hitGroupProgramGroupDescription, 1, // array
                                    &hitGroupProgramGroupOptions,
                                    logBuffer, &logBufferSize,
                                    &hitGroupProgramGroup_)
        );
        logBuffer[logBufferSize] = '\0';
        logInfo_('[' + whereInProgram.toString() + "] " + (char*) logBuffer);
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::makeMissProgramGroup_(char *logBuffer,
                                                 size_t logBufferSize,
                                                 ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    try {
        // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#ga0b44fd0708cced8b77665cfac5453573
        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_desc.html
        OptixProgramGroupDesc missProgramGroupDescription{};
        {
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gabca35b1218b4df575a5c42926da0d978
            missProgramGroupDescription.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
            // https://raytracing-docs.nvidia.com/optix7/api/group__optix__types.html#gaa65e764f5bba97fda45d4453c0464596
            missProgramGroupDescription.flags = OPTIX_PROGRAM_GROUP_FLAGS_NONE;

            // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_single_module.html
            missProgramGroupDescription.miss.module = optixModule_;
            missProgramGroupDescription.miss.entryFunctionName = "__miss__noop";
        }
        // https://raytracing-docs.nvidia.com/optix7/api/struct_optix_program_group_options.html
        OptixProgramGroupOptions missProgramGroupOptions{}; // no payload

        // https://raytracing-docs.nvidia.com/optix7/api/optix__host_8h.html#aa3515445a876a8a381ced002e4020d42
        OPTIX_CHECK(
            optixProgramGroupCreate(optixDeviceContext_,
                                    &missProgramGroupDescription, 1, // array
                                    &missProgramGroupOptions,
                                    logBuffer, &logBufferSize,
                                    &missProgramGroup_)
        );
        logBuffer[logBufferSize] = '\0';
        logInfo_('[' + whereInProgram.toString() + "] " + (char*) logBuffer);
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::buildOptixPipeline_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    try {
        ExecutionCursor whereInSubProgram = whereInProgram.nextLevel();

        // Char buffers for optix logging.
        size_t const kOptixLogBufferSize = 1024;
        char optixLogBuffer[kOptixLogBufferSize];
        optixLogBuffer[kOptixLogBufferSize-1] = '\0'; // ensure NULL-TERMINATED
        // ... Provide (kOptixLogBufferSize-1) as buffer-size in all uses to maintain this property.

        OptixPipelineCompileOptions optixPipelineCompileOptions =
            makeOptixPipelineCompileOptions_();
        makeOptixModule_(optixPipelineCompileOptions,
                        optixLogBuffer, kOptixLogBufferSize-1,
                        whereInSubProgram.advance());
        OptixPipelineLinkOptions optixPipelineLinkOptions =
            makeOptixPipelineLinkOptions_();
        makeRaygenProgramGroup_(optixLogBuffer, kOptixLogBufferSize-1, whereInSubProgram.advance());
        makeHitGroupProgramGroup_(optixLogBuffer, kOptixLogBufferSize-1, whereInSubProgram.advance());
        makeMissProgramGroup_(optixLogBuffer, kOptixLogBufferSize-1, whereInSubProgram.advance());

        constexpr unsigned int kNumProgramGroups = 3;
        std::array<OptixProgramGroup, kNumProgramGroups> optixProgramGroups{
            raygenProgramGroup_, hitGroupProgramGroup_, missProgramGroup_
        };
        
        size_t optixLogBufferSize = kOptixLogBufferSize-1;
        OPTIX_CHECK(
            optixPipelineCreate(optixDeviceContext_,
                                &optixPipelineCompileOptions,
                                &optixPipelineLinkOptions,
                                optixProgramGroups.data(), kNumProgramGroups,
                                optixLogBuffer, &optixLogBufferSize,
                                &optixPipeline_);
        );
        optixLogBuffer[optixLogBufferSize] = '\0';
        logInfo_('[' + whereInProgram.toString() + "] " + (char*) optixLogBuffer);
    }
    catch(std::exception const& e) {
        logInfo_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

template<typename T>
struct ShaderBindingTableRecord
{
    __align__(OPTIX_SBT_RECORD_ALIGNMENT)
        char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    T data;
};

struct RayGenData { /* TODO */};
using RayGenSbtRecord = ShaderBindingTableRecord<RayGenData>;

struct MissData { /* TODO */ };
using MissSbtRecord = ShaderBindingTableRecord<MissData>;

struct HitGroupData { /* TODO */ };
using HitGroupSbtRecord = ShaderBindingTableRecord<HitGroupData>;

void*
AssimpOptixRayTracer::makeRaygenSbtRecord_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    void* result { nullptr };
    try {
        RayGenSbtRecord raygenSbtRecord;
        OPTIX_CHECK(
            optixSbtRecordPackHeader(raygenProgramGroup_, (void*) &raygenSbtRecord)
        );
        // Populate raygenSbtRecord.data here if needed in future
        AsyncCudaBuffer raygenSbtRecordOnGpu{cudaStream_};
        raygenSbtRecordOnGpu.asyncAllocAndUpload(&raygenSbtRecord, 1);
        result = raygenSbtRecordOnGpu.detach();
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
    return result;
}

std::pair<void*, unsigned int>
AssimpOptixRayTracer::makeMissSbtRecord_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    std::pair<void*, unsigned int> result { 0, 0 };
    try {
        MissSbtRecord missSbtRecord;
        OPTIX_CHECK(
            optixSbtRecordPackHeader(missProgramGroup_, (void*) &missSbtRecord)
        );
        AsyncCudaBuffer missSbtRecordOnGpu{cudaStream_};
        missSbtRecordOnGpu.asyncAllocAndUpload(&missSbtRecord, 1);
        result.second = missSbtRecordOnGpu.sizeInBytes;
        // Destroys internal metadata, so ordered after extracting sizeInBytes.
        result.first = missSbtRecordOnGpu.detach();
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
    return result;
}

std::pair<void*, unsigned int>
AssimpOptixRayTracer::makeHitGroupSbtRecord_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger scopeLogger(logInfo_, whereInProgram, __func__);
    std::pair<void*, unsigned int> result{ 0, 0 };
    try {
        HitGroupSbtRecord hitGroupSbtRecord;
        OPTIX_CHECK(
            optixSbtRecordPackHeader(hitGroupProgramGroup_, (void*) &hitGroupSbtRecord)
        );
        AsyncCudaBuffer hitGroupSbtRecordOnGpu{cudaStream_};
        hitGroupSbtRecordOnGpu.asyncAllocAndUpload(&hitGroupSbtRecord, 1);
        result.second = hitGroupSbtRecordOnGpu.sizeInBytes;
        // Destroys internal metadata, so ordered after extracting sizeInBytes.
        result.first = hitGroupSbtRecordOnGpu.detach();
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
    return result;
}

/// @brief Populates optixSbt_
/// @param indent Indentation-prefix
void AssimpOptixRayTracer::makeOptixShaderBindingTable_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger raiiScopeLogger(logInfo_, whereInProgram, __func__);
    try {
        ExecutionCursor whereInSubProgram = whereInProgram.nextLevel();
        optixSbt_.raygenRecord = (hipDeviceptr_t) makeRaygenSbtRecord_(whereInSubProgram.advance());

        optixSbt_.missRecordCount = 1;
        std::pair<void*, unsigned int> missSbtRecordInfo = makeMissSbtRecord_(whereInSubProgram.advance());
        optixSbt_.missRecordBase = (hipDeviceptr_t) missSbtRecordInfo.first;
        optixSbt_.missRecordStrideInBytes = missSbtRecordInfo.second;

        optixSbt_.hitgroupRecordCount = 1;
        std::pair<void*, unsigned int> hitGroupSbtRecordInfo = makeHitGroupSbtRecord_(whereInSubProgram.advance());
        optixSbt_.hitgroupRecordBase = (hipDeviceptr_t) hitGroupSbtRecordInfo.first;
        optixSbt_.hitgroupRecordStrideInBytes = hitGroupSbtRecordInfo.second;

        optixSbt_.callablesRecordBase = 0;
        optixSbt_.callablesRecordCount = 0;
        optixSbt_.callablesRecordStrideInBytes = 0;

        optixSbt_.exceptionRecord = 0;
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

void AssimpOptixRayTracer::initializeOptix_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger raiiScopeLogger(logInfo_, whereInProgram, __func__);

	hipFree(0);
	cuCtx_ = 0;
    ExecutionCursor whereInSubProgram = whereInProgram.nextLevel();
	try {
		OPTIX_CHECK(optixInit());
		createOptixDeviceContext_(whereInSubProgram.advance()); // set optixDeviceContext_
        buildOptixPipeline_(whereInSubProgram.advance()); // set optixModule_, optixPipeline_
        makeOptixShaderBindingTable_(whereInSubProgram.advance()); // set optixSbt_
	}
	catch(std::exception& e) {
		logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
	}
}

void AssimpOptixRayTracer::finalizeOptix_(ExecutionCursor whereInProgram)
{
    RaiiScopeLimitsLogger raiiScopeLogger(logInfo_, whereInProgram, __func__);
    try {
        OPTIX_CHECK( optixProgramGroupDestroy(raygenProgramGroup_) );
        OPTIX_CHECK( optixProgramGroupDestroy(hitGroupProgramGroup_) );
        OPTIX_CHECK( optixProgramGroupDestroy(missProgramGroup_) );
        OPTIX_CHECK( optixPipelineDestroy(optixPipeline_) );
        OPTIX_CHECK( optixModuleDestroy(optixModule_) );
        OPTIX_CHECK( optixDeviceContextDestroy(optixDeviceContext_) );
    }
    catch(std::exception const& e) {
        logError_('[' + whereInProgram.toString() + "] ***ERROR*** " + e.what());
    }
}

} // namespace manojr
